#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <chrono>
typedef std::chrono::high_resolution_clock Clock;

using namespace std;


__global__ void VectorAdd(int *res, int *op1, int *op2, int n)
{
	//int i = blockIdx.x;
	int i = threadIdx.x;
	//for (int i = 0; i < n; i++)
	if (i < n)
	{
		res[i] = op1[i] + op2[i];
	}
}

int main()
{
	const int N = 1000;//32000000;
	int *a, *b, *c;

	//a = new int[N];
	//b = new int[N];
	//c = new int[N];
	hipMallocManaged(&a, N * sizeof(int));
	hipMallocManaged(&b, N * sizeof(int));
	hipMallocManaged(&c, N * sizeof(int));

	for (int i = 0; i < N; i++)
	{
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	auto t1 = Clock::now();
	//VectorAdd(c, a, b, N);
	VectorAdd << <1, N >> > (c, a, b, N);
	auto t2 = Clock::now();
	hipDeviceSynchronize();


	for (int i = 0; i < N; i++)
	{
		cout << c[i] << endl;
	}

	std::cout << "Time: "
		<< std::chrono::duration_cast<std::chrono::nanoseconds>(t2 - t1).count()
		<< " nanoseconds" << std::endl;

	//delete[] a;
	//delete[] b;
	//delete[] c;
	hipFree(a);
	hipFree(b);
	hipFree(c);

	//system("Pause");
	return 0;
}