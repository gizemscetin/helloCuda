#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <chrono>
typedef std::chrono::high_resolution_clock Clock;

using namespace std;

__global__ void VectorMult(int *res, int *op1, int *op2, int n)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (; i < n; i += step)
	{
		res[i] = op1[i] * op2[i];
	}
}


__global__ void VectorAdd(int *res, int *op1, int *op2, int n)
{
	//int i = threadIdx.x;
	//int step = blockDim.x;
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int step = blockDim.x * gridDim.x;
	for (; i < n; i+=step)
	{
		res[i] = op1[i] + op2[i];
	}
}

int main()
{
	const int N = 1000000;//1048576;//32000000;
	int *a, *b, *c;

	//a = new int[N];
	//b = new int[N];
	//c = new int[N];
	hipMallocManaged(&a, N * sizeof(int));
	hipMallocManaged(&b, N * sizeof(int));
	hipMallocManaged(&c, N * sizeof(int));

	for (int i = 0; i < N; i++)
	{
		a[i] = rand() % 100;
		b[i] = rand() % 100;
	}

	auto t1 = Clock::now();
	//VectorAdd(c, a, b, N);
	int blockSize = 1024;
	int numBlocks = (N + blockSize - 1) / blockSize;
	//VectorAdd <<<numBlocks, blockSize >>> (c, a, b, N);
	VectorMult << <numBlocks, blockSize >> > (c, a, b, N);
	auto t2 = Clock::now();
	hipDeviceSynchronize();


	for (int i = N-5; i < N; i++)
	{
		cout << c[i] << endl;
	}

	std::cout << "Time: "
		<< std::chrono::duration_cast<std::chrono::milliseconds>(t2 - t1).count()
		<< " milliseconds" << std::endl;

	//delete[] a;
	//delete[] b;
	//delete[] c;
	hipFree(a);
	hipFree(b);
	hipFree(c);

	//system("Pause");
	return 0;
}